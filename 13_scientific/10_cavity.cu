#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <hip/hip_runtime.h>

#define NX 41
#define NY 41
#define NT 500
#define NIT 50
#define DX 2.0 / (NX - 1)
#define DY 2.0 / (NY - 1)
#define DT 0.01
#define RHO 1.0
#define NU 0.02

__global__ void compute_b(float* b, float* u, float* v, double dt, double dx, double dy, double rho) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = j * NX + i;

    if (j > 0 && j < NY - 1 && i > 0 && i < NX - 1) {
        b[idx] = rho * (1.0 / dt *
                        ((u[idx + 1] - u[idx - 1]) / (2.0 * dx) +
                         (v[idx + NX] - v[idx - NX]) / (2.0 * dy)) -
                        pow((u[idx + 1] - u[idx - 1]) / (2.0 * dx), 2) -
                        2.0 * ((u[idx + NX] - u[idx - NX]) / (2.0 * dy) * (v[idx + 1] - v[idx - 1]) / (2.0 * dx)) -
                        pow((v[idx + NX] - v[idx - NX]) / (2.0 * dy), 2));
    }
}

__global__ void pressure_poisson(float* p, float* pn, float* b, double dx, double dy) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = j * NX + i;

    if (j > 0 && j < NY - 1 && i > 0 && i < NX - 1) {
        p[idx] = ((pn[idx + 1] + pn[idx - 1]) * dy * dy +
                  (pn[idx + NX] + pn[idx - NX]) * dx * dx) /
                 (2 * (dx * dx + dy * dy)) -
                 dx * dx * dy * dy / (2 * (dx * dx + dy * dy)) * b[idx];
    }

    // Enforce boundary conditions for pressure
    if (i == NX - 1) {
        p[idx] = p[idx - 1]; // dp/dx = 0 at x = 2
    }
    if (i == 0) {
        p[idx] = p[idx + 1]; // dp/dx = 0 at x = 0
    }
    if (j == NY - 1) {
        p[idx] = 0; // p = 0 at y = 2
    }
    if (j == 0) {
        p[idx] = p[idx + NX]; // dp/dy = 0 at y = 0
    }
}

__global__ void update_velocity(float* u, float* v, float* un, float* vn, float* p, double dt, double dx, double dy, double rho, double nu) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = j * NX + i;

    if (j > 0 && j < NY - 1 && i > 0 && i < NX - 1) {
        u[idx] = un[idx] - un[idx] * dt / dx * (un[idx] - un[idx - 1]) -
                 vn[idx] * dt / dy * (un[idx] - un[idx - NX]) -
                 dt / (2 * rho * dx) * (p[idx + 1] - p[idx]) +
                 nu * (dt / (dx * dx) * (un[idx + 1] - 2 * un[idx] + un[idx - 1]) +
                       dt / (dy * dy) * (un[idx + NX] - 2 * un[idx] + un[idx - NX]));

        v[idx] = vn[idx] - un[idx] * dt / dx * (vn[idx] - vn[idx - 1]) -
                 vn[idx] * dt / dy * (vn[idx] - vn[idx - NX]) -
                 dt / (2 * rho * dy) * (p[idx + NX] - p[idx]) +
                 nu * (dt / (dx * dx) * (vn[idx + 1] - 2 * vn[idx] + vn[idx - 1]) +
                       dt / (dy * dy) * (vn[idx + NX] - 2 * vn[idx] + vn[idx - NX]));
    }

    // Enforce boundary conditions for velocity
    if (j == 0 || j == NY - 1 || i == 0 || i == NX - 1) {
        u[idx] = 0;
        v[idx] = 0;
    }

    if (j == NY - 1) {
        u[idx] = 1; // velocity boundary condition at the top (moving lid)
    }
}

int main() {
    // Allocate and initialize host arrays
    float *h_u, *h_v, *h_p, *h_b, *h_un, *h_vn, *h_pn;
    size_t size = NX * NY * sizeof(float);

    h_u = (float *)malloc(size);
    h_v = (float *)malloc(size);
    h_p = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_un = (float *)malloc(size);
    h_vn = (float *)malloc(size);
    h_pn = (float *)malloc(size);

    for (int j = 0; j < NY; j++) {
        for (int i = 0; i < NX; i++) {
            int idx = j * NX + i;
            h_u[idx] = 0.0;
            h_v[idx] = 0.0;
            h_p[idx] = 0.0;
            h_b[idx] = 0.0;
        }
    }

    // Allocate device arrays
    float *d_u, *d_v, *d_p, *d_b, *d_un, *d_vn, *d_pn;
    hipMalloc((void **)&d_u, size);
    hipMalloc((void **)&d_v, size);
    hipMalloc((void **)&d_p, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_un, size);
    hipMalloc((void **)&d_vn, size);
    hipMalloc((void **)&d_pn, size);

    // Copy host arrays to device
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
    hipMemcpy(d_p, h_p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Write results to files
    std::ofstream ufile("u.dat");
    std::ofstream vfile("v.dat");
    std::ofstream pfile("p.dat");

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((NX + blockSize.x - 1) / blockSize.x, (NY + blockSize.y - 1) / blockSize.y);

    for (int n = 0; n < NT; n++) {
        compute_b<<<gridSize, blockSize>>>(d_b, d_u, d_v, DT, DX, DY, RHO);

        for (int it = 0; it < NIT; it++) {
            // Copy p to pn
            hipMemcpy(d_pn, d_p, size, hipMemcpyDeviceToDevice);
            pressure_poisson<<<gridSize, blockSize>>>(d_p, d_pn, d_b, DX, DY);
            hipDeviceSynchronize(); // Ensure pressure_poisson kernel is completed
        }

        // Copy u and v to un and vn
        hipMemcpy(d_un, d_u, size, hipMemcpyDeviceToDevice);
        hipMemcpy(d_vn, d_v, size, hipMemcpyDeviceToDevice);

        update_velocity<<<gridSize, blockSize>>>(d_u, d_v, d_un, d_vn, d_p, DT, DX, DY, RHO, NU);
        hipDeviceSynchronize(); // Ensure update_velocity kernel is completed

        // Apply boundary conditions on the host
        hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_p, d_p, size, hipMemcpyDeviceToHost);

        for (int j=0; j<NY; j++) {
            // Compute u[j][0], u[j][nx-1], v[j][0], v[j][nx-1]
            h_u[j*NY+0] = 0;
            h_u[j*NY+(NX-1)]= 0;
            h_v[j*NY+0] = 0;
            h_v[j*NY+(NX-1)]=0;
        }
        for (int i=0; i<NX; i++) {
            // Compute u[0][i], u[ny-1][i], v[0][i], v[ny-1][i]
            h_u[0*NY+i] = 0;
            h_u[(NX-1)*NY+i] = 1;
            h_v[0*NY+i]=0;
            h_v[(NY-1)*NY+i]=0;
        }
        if (n % 10 == 0) {
            for (int j=0; j<NY; j++)
                for (int i=0; i<NX; i++)
                    ufile << h_u[j*NY+ i] << " ";
            ufile << "\n";
            for (int j=0; j<NY; j++)
                for (int i=0; i<NX; i++)
                    vfile << h_v[j*NY+i] << " ";
            vfile << "\n";
            for (int j=0; j<NY; j++)
                for (int i=0; i<NX; i++)
                    pfile << h_p[j*NY+i] << " ";
            pfile << "\n";
        }
    }

    ufile.close();
    vfile.close();
    pfile.close();

    // Free device memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_p);
    hipFree(d_b);
    hipFree(d_un);
    hipFree(d_vn);
    hipFree(d_pn);

    // Free host memory
    free(h_u);
    free(h_v);
    free(h_p);
    free(h_b);
    free(h_un);
    free(h_vn);
    free(h_pn);

    return 0;
}
